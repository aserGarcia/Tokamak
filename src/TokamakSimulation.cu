#include "hip/hip_runtime.h"
// Optimized using shared memory and on chip memory
// Compile source: $- nvcc src/TokamakSimulation.cu -o nBody -lglut -lm -lGLU -lGL
// Run Executable: $- ./nBody
//To stop hit "control c" in the window you launched it from.
//Make movies https://gist.github.com/JPEGtheDev/db078e1b066543ce40580060eee9c1bf
#include <GL/freeglut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../includes/cudaErrCheck.cuh"

#define NR_NEUTRONS 8 
#define NR_ELECTRONS 8 
#define NR_PROTONS 8

//atomic mass (u)
#define MASS_PROTON 1.007276 
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj" //256 vertices-shape (for array simplicity)
#define N 16*16*16

//***********************
// TODO: 
//		Check units velocity calculation mag
//		ಠ_ಠ
//***********************

// Globals
float4 *p;
float3 *v, *f, *reactor;

//DeviceStruct stores GPU(s) info//
struct DeviceStruct {
	int deviceID;
	int offset;
};

void read_obj(){
	FILE *fp = fopen(PATH, "r");
	char c, line[256];
	memset(line, 0, 256);
	ERROR_CHECK( hipMallocManaged(&reactor, SHAPE_SIZE*SHAPE_CT*sizeof(float3)) );

    int j =0;
    while(fgets(line, sizeof(line), fp) != 0){
		c = line[0];
        if(c=='v'){
			sscanf(line, "%c %f %f %f\n", &c, &reactor[j].x, &reactor[j].y, &reactor[j].z);
			j++;
		}
    }
	fclose(fp);
}

void set_initial_conditions(){

	ERROR_CHECK( hipMallocManaged(&p, N*sizeof(float4)) );
	ERROR_CHECK( hipMallocManaged(&v, N*sizeof(float3)) );
	ERROR_CHECK( hipMallocManaged(&f, N*sizeof(float3)) );

	float numc = 1.0;
	int separation = 360*8/N;
	int nr_circles = N/16;
	float r = 5.0;
	for(int num=0;num<N;num++){
		p[num].x = r*cos(separation*num);
		p[num].y = numc;
		p[num].z = r*sin(separation*num);
		p[num].w = MASS_PROTON;
		
		v[num].x = -1.5*p[num].x;
		v[num].y = 0.0;
		v[num].z = 1.5*sqrtf(r*r-p[num].x*p[num].x);

		f[num].x = 0.0;
		f[num].y = 0.0;
		f[num].z = 0.0;
		
		if(num%nr_circles==0){
			numc += 0.2;
		}
	}
}


void draw_picture(){

	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	glColor3d(0.6,0.8,1.0);
	glPointSize(3.0);
	glBegin(GL_POINTS);
	for(int i=0; i<N; i++)
	{
		glVertex3f(p[i].x, p[i].y, p[i].z);
	}
	glEnd();
	
	//drawing ractor
	glColor3d(1.0,0.0,0.0);
	glPointSize(5.0);
	glBegin(GL_POINTS);
	for(int i=0; i<SHAPE_SIZE*SHAPE_CT; i++)
	{
		glVertex3f(reactor[i].x, reactor[i].y, reactor[i].z);
	}
	glEnd();
	glutSwapBuffers();
}

__device__ float3 getBodyBodyForce(float4 p0, float4 p1){
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz;
	float inv_r = 1/sqrt(r2);
	
    float force  = (G*p0.w*p1.w)/(r2);// - (H*p0.w*p1.w)/(r2*r2);
    
    f.x = force*dx*inv_r;
    f.y = force*dy*inv_r;
    f.z = force*dz*inv_r;
    
    return(f);
}

__device__ float3 getMagForce(float4 p0, float3 v0, float3 dl_tail, float3 dl_head, float I){
	//dl is the section of wire
	float3 dB, dl;
	dl.x = dl_head.x-dl_tail.x;
	dl.y = dl_head.y-dl_tail.y;
	dl.z = dl_head.z-dl_tail.z;

	float rx = p0.x-dl_tail.x;
    float ry = p0.y-dl_tail.y;
	float rz = p0.z-dl_tail.z;

	float r2 = rx*rx+ry*ry+rz*rz;
	float inv_r2 = 1/r2;
	float inv_r = 1/sqrtf(r2);
	float3 rhat = {rx*inv_r, ry*inv_r, rz*inv_r};

	//(dl cross rhat)/r2 = force
	//gamma is mu0*I/4Pi which simplifies to Ie-7
	float gamma = I;
	dB.x = gamma*(dl.y*rhat.z-dl.z*rhat.y)*inv_r2;
	dB.y = gamma*(dl.z*rhat.x-dl.x*rhat.z)*inv_r2;
	dB.z = gamma*(dl.x*rhat.y-dl.y*rhat.x)*inv_r2;

	return (dB);
}

__global__ void getForcesMag(float4 *pos, float3 *vel, float3 *force, float3 *reactor, int offset){
	
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	float3 total_force, B, dB, dl_tail, dl_head, velMe;
	float4 posMe;
	__shared__ float3 shared_r[BLOCK];

	total_force.x = B.x = 0.0;
	total_force.y = B.y = 0.0;
	total_force.z = B.z = 0.0;

	posMe.x = pos[id+offset].x;
	posMe.y = pos[id+offset].y;
	posMe.z = pos[id+offset].z;
	posMe.w = pos[id+offset].w;

	velMe.x = vel[id+offset].x;
	velMe.y = vel[id+offset].y;
	velMe.z = vel[id+offset].z;
	
	for(int k=0;k<SHAPE_CT;k++){
		shared_r[threadIdx.x] = reactor[threadIdx.x + blockDim.x*k];
		__syncthreads();
		
		for(int j = 1; j<=SHAPE_SIZE; j++){
			dl_tail = shared_r[(j-1)];
			dl_head = shared_r[(j%SHAPE_SIZE)];
			dB = getMagForce(posMe, velMe, dl_tail, dl_head, 1.0); //current[i] =1
			
			B.x += dB.x;
			B.y += dB.y;
			B.z += dB.z;
		}
	}

	total_force.x = (velMe.y*B.z-velMe.z*B.y);
	total_force.y = (velMe.z*B.x-velMe.x*B.z);
	total_force.z = (velMe.x*B.y-velMe.y*B.x);

	if(id<N){
		force[id+offset].x += total_force.x;
		force[id+offset].y += total_force.y;
		force[id+offset].z += total_force.z;
	}
}

__global__ void getForces(float4 *g_pos, float3 *force, int offset, int device_ct){
	int ii;
    float3 force_b2b, forceSum;
    float4 posMe;
    __shared__ float4 shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	forceSum.z = 0.0;
		
	posMe.x = g_pos[id+offset].x;
	posMe.y = g_pos[id+offset].y;
	posMe.z = g_pos[id+offset].z;
	posMe.w = g_pos[id+offset].w;
	    
    for(int j=0; j < gridDim.x*device_ct; j++)
    {
    	shPos[threadIdx.x] = g_pos[threadIdx.x + blockDim.x*j];
    	__syncthreads();
   
		#pragma unroll 32
        for(int i=0; i < blockDim.x; i++)	
        {
        	ii = i + blockDim.x*j;
		    if(ii != id+offset && ii < N) 
		    {
		    	force_b2b = getBodyBodyForce(posMe, shPos[i]);
			    forceSum.x += force_b2b.x;
			    forceSum.y += force_b2b.y;
			    forceSum.z += force_b2b.z;
		    }
	   	}
	}

	if(id <N){
	    force[id+offset].x = forceSum.x;
	    force[id+offset].y = forceSum.y;
	    force[id+offset].z = forceSum.z;
    }
}

__global__ void moveBodies(float4 *pos, float3 *vel, float3 *force, int offset){
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    if(id < N){
		float inv_mass = 1/pos[id+offset].w;
	    vel[id+offset].x += ((force[id+offset].x-DAMP*vel[id+offset].x)*inv_mass)*DT;
	    vel[id+offset].y += ((force[id+offset].y-DAMP*vel[id+offset].y)*inv_mass)*DT;
		vel[id+offset].z += ((force[id+offset].z-DAMP*vel[id+offset].z)*inv_mass)*DT;
		
		pos[id+offset].x += vel[id+offset].x*DT;
		pos[id+offset].y += vel[id+offset].y*DT;
		pos[id+offset].z += vel[id+offset].z*DT;
    }
}

void n_body(){
	int deviceCount;
	ERROR_CHECK( hipGetDeviceCount ( &deviceCount ) );
	DeviceStruct* dev = (DeviceStruct*)malloc(deviceCount*sizeof(DeviceStruct));
	
	for(int i = 0; i<deviceCount; i++){
		dev[i].deviceID = i;
		dev[i].offset = i*N/deviceCount;
	}

	dim3 block(BLOCK);
	dim3 grid((N/deviceCount - 1)/BLOCK + 1);
	
	float dt;
	int   tdraw = 0; 
	float time = 0.0;
	float elapsedTime;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	dt = DT;
	while(time < STOP_TIME){	
		for(int i = 0; i < deviceCount; i++){
			getForces<<<grid, block>>>(p, f, dev[i].offset, deviceCount);
			ERROR_CHECK( hipPeekAtLastError() );

			getForcesMag<<<grid,block>>>(p, v, f, reactor, dev[i].offset);
			ERROR_CHECK( hipPeekAtLastError() );

			moveBodies<<<grid, block>>>(p, v, f, dev[i].offset);
			ERROR_CHECK( hipPeekAtLastError() );
			hipDeviceSynchronize();
		}

		//To kill the draw comment out the next 7 lines.
		if(tdraw == DRAW){
			draw_picture();
			//break the for loop by closing window
			glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_GLUTMAINLOOP_RETURNS);
			glutMainLoopEvent();
			if(!glutGetWindow()){ break; }
			tdraw = 0;
		}
		tdraw++;
		time += dt;
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\n\nGPU time = %3.1f seconds\n", elapsedTime/1000.0);

	hipDeviceSynchronize();
	hipFree(p);
	hipFree(v);
	hipFree(f);
	hipFree(reactor);
}

void control(){	
	read_obj();
	set_initial_conditions();
	draw_picture();
    n_body();
	printf("\n DONE \n");
}

void Display(void){
	gluLookAt(EYE, EYE, EYE, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	control();
}

void reshape(int w, int h){
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, FAR);
	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv){
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("2 Body 3D");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.2, 0.2, 0.2, 1.0};
	GLfloat light_diffuse[]  = {0.8, 0.8, 0.8, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoopEvent();
	return 0;
}
